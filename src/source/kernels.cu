#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <chrono>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>

__global__ void
setup_curand(
    hiprandState* state,
    const unsigned long time
);

__global__ void
kernel_alter(
    unsigned long n,
    unsigned char* contents,
    unsigned char min_deviation,
    unsigned char max_deviation,
    hiprandState* state
);

__global__ void
kernel_pass(
    unsigned long n,
    unsigned char* contents,
    float chance,
    hiprandState* state
);


namespace CUDA::kernels
{

    int call_alter_kernel(
        unsigned char* cpu_contents,
        unsigned long contents_size,
        unsigned char min_deviation,
        unsigned char max_deviation
    )
    {
        auto current_time_count = std::chrono::high_resolution_clock::now().time_since_epoch().count();

        int thread_count = 2 << 9;
        int block_count = (contents_size + thread_count - 1) / thread_count;

        hiprandState* random_states;
        if (hipMalloc(&random_states, sizeof(hiprandState) * thread_count) != hipSuccess)
            return 1;

        printf("Setting up the random states...\n");
        ::setup_curand<<<1, thread_count>>>(random_states, current_time_count);

        printf("Altering the given data...\n");
        ::kernel_alter<<<block_count, thread_count>>>(contents_size, cpu_contents, min_deviation, max_deviation, random_states);
        printf("Waiting on device synchronization...\n");
        hipDeviceSynchronize();

        return 0;
    }

    int call_pass_kernel(
        unsigned char* cpu_contents,
        unsigned long contents_size,
        float chance
    )
    {
        auto current_time_count = std::chrono::high_resolution_clock::now().time_since_epoch().count();

        int thread_count = 2 << 9;
        int block_count = (contents_size + thread_count - 1) / thread_count;

        hiprandState* random_states;
        if (hipMalloc(&random_states, sizeof(hiprandState) * thread_count) != hipSuccess)
            return 1;

        printf("Setting up the random states...\n");
        ::setup_curand<<<1, thread_count>>>(random_states, current_time_count);

        printf("Altering the given data...\n");
        ::kernel_pass<<<block_count, thread_count>>>(contents_size, cpu_contents, chance, random_states);

        printf("Waiting on device synchronization...\n");
        hipDeviceSynchronize();

        return 0;
    }

} // namespace CUDA::kernels


__global__ void
setup_curand(
    hiprandState* state,
    const unsigned long time
)
{
    hiprand_init(time, threadIdx.x, 0, state + threadIdx.x);
}

__global__ void
kernel_alter(
    unsigned long n,
    unsigned char* contents,
    unsigned char min_deviation,
    unsigned char max_deviation,
    hiprandState* state
)
{
    hiprandState currentRandomState = state[threadIdx.x];

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (long i = idx; i < n; i += stride)
    {
        unsigned char deviation = (unsigned char)(min_deviation + hiprand_uniform(&currentRandomState) * (max_deviation - min_deviation));

        contents[i] += deviation;
    }
}

__global__ void
kernel_pass(
    unsigned long n,
    unsigned char* contents,
    float chance,
    hiprandState* state
)
{
    hiprandState currentRandomState = state[threadIdx.x];

    unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned long stride = blockDim.x * gridDim.x;

    for (unsigned long i = idx; i < n; i += stride)
    {
        float random_value = hiprand_uniform(&currentRandomState) * 100.f;
        if (random_value <= chance)
            contents[i] = (unsigned char)(hiprand_uniform(&currentRandomState) * 255.f);
    }
}
